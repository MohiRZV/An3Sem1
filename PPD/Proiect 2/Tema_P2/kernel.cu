#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <random>
#include <time.h>
#include <fstream>
#include <strstream>
#include <string>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

using std::cout;
using std::endl;
using std::ifstream;
using std::ofstream;
using std::string;

#define THREADS_NO 512


hipError_t blurImageWithCuda(unsigned char* matrix, unsigned char* result_matrix, float* filter, int rows, int columns);


__global__ void blurImageKernel(unsigned char* matrix, unsigned char* result, float* filter, int rows, int columns, int filter_rows, int filter_columns)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < rows * columns) {
        int row = index / columns;
        int column = index % columns;

        int limSup = row - filter_rows / 2;
        int limInf = row + filter_rows / 2;
        int limStg = column - filter_columns / 2;
        int limDr = column + filter_columns / 2;

        int linieStartFiltru = 0;
        int linieStopFiltru = filter_rows;
        int colStartFiltru = 0;
        int colStopFiltru = filter_columns;

        if (limSup < 0) {
            //depasire limita superioara
            linieStartFiltru = 0 - limSup;
            limSup = 0;
        }
        if (limInf > rows - 1) {
            //depasire limita inferioara
            linieStopFiltru = filter_rows - (limInf - rows + 1);
            limInf = rows - 1;
        }
        if (limStg < 0) {
            //depasire limita stanga
            colStartFiltru = 0 - limStg;
            limStg = 0;
        }

        if (limDr > columns - 1) {
            //depasire limita dreapta
            colStopFiltru = filter_columns - (limDr - columns + 1);
            limDr = columns - 1;
        }

        int linieFiltru = linieStartFiltru, colFiltru = colStartFiltru;
        char sum = 0;
        for (int i = limSup; i <= limInf; i++) {
            for (int j = limStg; j <= limDr; j++) {
                sum += matrix[i * columns + j] * filter[linieFiltru * filter_columns + colFiltru];
                colFiltru++;
            }
            linieFiltru++;
            colFiltru = colStartFiltru;
        }
        result[row * rows + column] = sum / (filter_columns * filter_rows);
    }

}


void filter_gaussian_init(float* filter) {
    filter[0] = 1;
    filter[1] = 1;
    filter[2] = 1;
    filter[3] = 1;
    filter[4] = 1;
    filter[5] = 1;
    filter[6] = 1;
    filter[7] = 1;
    filter[8] = 1;
}


int main()
{


    cv::Mat inputImageRGBA;
    cv::Mat outputImageRGBA;

    uchar4* inputImageRGBAMatrix;
    uchar4* outputImageRGBAMatrix;

    string input_file{ "portrait.jpg" };
    string output_file{ "portrait_blured.jpg" };
    cv::cvtColor(cv::imread(input_file.c_str(),1), inputImageRGBA, 2);

    if (inputImageRGBA.empty()) {
        std::cerr << "Couldn't open file: " << input_file << std::endl;
        exit(1);
    }

    int numRows = inputImageRGBA.rows;
    int numCols = inputImageRGBA.cols;

    inputImageRGBAMatrix = new uchar4[numRows * numCols];
    outputImageRGBAMatrix = new uchar4[numRows * numCols];
    
    outputImageRGBA.create(numRows, numCols, CV_8UC4);

    memcpy(inputImageRGBAMatrix, (uchar4*)inputImageRGBA.ptr<unsigned char>(0), numRows*numCols*sizeof(uchar4));
    
    const size_t numPixels = numRows * numCols ;

    unsigned char* red = new unsigned char[numPixels];
    unsigned char* blue = new unsigned char[numPixels];
    unsigned char* green = new unsigned char[numPixels];

    unsigned char* redBlurred = new unsigned char[numPixels];
    unsigned char* blueBlurred = new unsigned char[numPixels];
    unsigned char* greenBlurred = new unsigned char[numPixels];

    float* filter = new float[3 * 3];

    filter_gaussian_init(filter);

    
    for (size_t i = 0; i < numRows * numCols; ++i) {
        uchar4 rgba = inputImageRGBAMatrix[i];
        red[i] = rgba.x;
        green[i] = rgba.y;
        blue[i] = rgba.z;
    }


    hipError_t cudaStatus = blurImageWithCuda(red, redBlurred, filter, numRows, numCols);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "blurWithCuda failed!");
        return 1;
    }
    cudaStatus = blurImageWithCuda(green, greenBlurred, filter, numRows, numCols);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "blurWithCuda failed!");
        return 1;
    }
    cudaStatus = blurImageWithCuda(blue, blueBlurred, filter, numRows, numCols);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "blurWithCuda failed!");
        return 1;
    }

    for (size_t i = 0; i < numRows * numCols; ++i) {
        uchar4 rgba = make_uchar4(redBlurred[i], greenBlurred[i], blueBlurred[i], 255);
        outputImageRGBAMatrix[i] = rgba;
    }

    memcpy((uchar4*)outputImageRGBA.ptr<unsigned char>(0), outputImageRGBAMatrix, numRows*numCols*sizeof(uchar4));

    cv::Mat imageOutputRGB;
    cv::cvtColor(outputImageRGBA, imageOutputRGB, 3);
    cv::imwrite(output_file.c_str(), imageOutputRGB);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    delete[] red;
    delete[] green;
    delete[] blue;

    delete[] redBlurred;
    delete[] greenBlurred;
    delete[] blueBlurred;
    
    
    return 0;
}


hipError_t blurImageWithCuda(unsigned char* matrix, unsigned char* result_matrix, float* filter, int rows, int columns)
{
    unsigned char* dev_matrix = 0;
    unsigned char* dev_result = 0;
    float* dev_filter = 0;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_result, rows * columns * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_matrix, rows * columns * sizeof(unsigned char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_filter, 3 * 3 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_matrix, matrix, rows * columns * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_result, result_matrix, rows * columns * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_filter, filter, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

    int blocksNo = rows * columns / (THREADS_NO - 1);
    blurImageKernel <<<blocksNo, THREADS_NO>>> (dev_matrix, dev_result, dev_filter, rows, columns, 3, 3);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "blurKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(result_matrix, dev_result, rows * columns * sizeof(unsigned char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_matrix);
    hipFree(dev_result);
    hipFree(dev_filter);

    return cudaStatus;

}